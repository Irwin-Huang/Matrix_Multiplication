#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_complex.h>
#include <stdio.h>
#include "kernel.h"
#define TX 32
#define TY 32


__global__
void matrixMult(hipDoubleComplex *d_A, hipDoubleComplex *d_B, hipDoubleComplex *d_C, int N)
{
	const int idx = threadIdx.x + blockDim.x*blockIdx.x;
	const int idy = threadIdx.y + blockDim.y*blockIdx.y;
	
	if ((idx >= N) || (idy >= N)) return;

	hipDoubleComplex tempProd = make_hipDoubleComplex(0,0);
	for (int i = 0;i < N; i++) 
	{
		tempProd = hipCmul(d_A[N*idy + i], d_B[N*i + idx]);
	}

	d_C[N*idy+idx] = tempProd;
}

void matrixLaunch(hipDoubleComplex *d_A, hipDoubleComplex *d_B, hipDoubleComplex *d_C, int N)
{
	const dim3 blockSize(TX, TY);
	const dim3 gridSize((N + TX - 1) / TX, (N + TY - 1) / TY);
	matrixMult << <gridSize, blockSize >> > (d_A, d_B, d_C, N);

}

__global__
void transposeMat(hipDoubleComplex *d_Q, hipDoubleComplex *d_P, int N)
{
	const int idx = threadIdx.x + blockDim.x*blockIdx.x;
	const int idy = threadIdx.y + blockDim.y*blockIdx.y;
	
	if ((idx >= N) || (idy >= N)) return;

	d_Q[N*idy + idx] = hipConj(d_P[N*idx + idy]);

}

void transposeLaunch(hipDoubleComplex *d_Q, hipDoubleComplex *d_P, int N)
{
	const dim3 blockSize(TX, TY);
	const dim3 gridSize((N + TX - 1) / TX, (N + TY - 1) / TY);
	transposeMat << <gridSize, blockSize >> > (d_Q, d_P, N);

}